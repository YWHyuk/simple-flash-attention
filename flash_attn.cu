#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Implement flash_attention kernel below
__global__ void _flash_attention() {}

torch::Tensor flash_attention(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    const int B = Q.size(0);	// Batch size
    const int nh = Q.size(1);	// Number of heads
    const int N = Q.size(2);	// Sequence size
    const int d = Q.size(3);	// Embedding size

    // Initialize O, l, m to HBM
    auto O = torch::zeros_like(Q);
    auto l = torch::zeros({B, nh, N});
    auto m = torch::full({B, nh, N}, -INFINITY);
    torch::Device device(torch::kCUDA);
    l = l.to(device); m = m.to(device);

    // Calculate SRAM size needed per block
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d\n", max_sram_size);

    // ============= Call kernel ==================
    // _flash_attention<<>>();
    // =============================================

    // Return output
    return O;
}
